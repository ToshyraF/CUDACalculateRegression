
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 16
__device__ float sum(float *input){
	float sums=0;
	for(int i=0;i<N;i++){
		sums += input[i];
	}
	return sums;
}
__device__ float sum_of_power(float *input){
	float sums=0;
	for(int i=0;i<N;i++){
		sums += input[i]*input[i];
	}
	return sums;
}
__device__ float sum_of_mul(float *input1,float *input2){
	float sums=0;
	for(int i=0;i<N;i++){
		sums += input1[i]*input2[i];
	}
	return sums;
}
__device__ float mean(float *input1){
	float sums = sum(input1);
	float means = sums/N;
	return means;
}
__device__ float findSlope(float *input1,float *input2){
	float sumX,sumY,sum_mulXY,meanX,sum_powerX;
	sumX = sum(input1);
	sumY = sum(input2);
	sum_mulXY = sum_of_mul(input1,input2);
	sum_powerX = sum_of_power(input1);
	meanX = mean(input1);
	float b = float(sum_mulXY - float((sumX*sumY)/N))/(sum_powerX-float(N*(meanX*meanX)));
	return b;
}
__device__ float cut_point_Y(float *input1,float *input2,float b){
	float meanY = mean(input2);
	float meanX = mean(input1);
	float a = meanY - (meanX*b);
	return a;
}
__global__ void regression(float *input1,float *input2,float *out_a,float *out_b){
	// int tid = blockDim.x*blockIdx.x + threadIdx.x;
	*out_b = findSlope(input1,input2);
	*out_a = cut_point_Y(input1,input2,findSlope(input1,input2));
		// *output = input[tid];
		//  __syncthreads();
	// }
}
int main(){


	float x[] = {3.5,3,3.2,3.1,3.6,3.9,3.4,3.4,2.9,3.1,3.7,3.4,3,4,4.4,3.9,3.5,3.8,3.8,3.4,3.7,3.6,3.3,3.4,3,3.4,3.5,3.4,3.2,3.1,3.4,4.1,4.2,3.1,3.2,3.5,3.6,3,3.4,3.5,2.3,3.2,3.5,3.8,3,3.8,3.7};
	float y[] = {5.1,4.9,4.7,4.6,5,5.4,4.6,5,4.4,4.9,5.4,4.8,4.3,5.8,5.7,5.4,5.1,5.7,5.1,5.4,5.1,4.6,5.1,4.8,5,5,5.2,5.2,4.7,4.8,5.4,5.2,5.5,4.9,5,5.5,4.9,4.4,5.1,5,4.5,4.4,5,5.1,4.8,4.6,5.3};
	
	float b,a;
	float *d_x,*d_y,*out_a,*out_b;

	size_t size = N*sizeof(float);

	hipMalloc(&d_x,size);
	hipMalloc(&d_y,size);

	hipMalloc(&out_a,sizeof(float));
	hipMalloc(&out_b,sizeof(float));

	hipMemcpy(d_x,x,size,hipMemcpyHostToDevice);
	hipMemcpy(d_y,y,size,hipMemcpyHostToDevice);

	regression<<<1,N>>>(d_x,d_y,out_a,out_b);

	hipMemcpy(&a,out_a,sizeof(float),hipMemcpyDeviceToHost);
	hipMemcpy(&b,out_b,sizeof(float),hipMemcpyDeviceToHost);

	printf("%f \n", a+(b*3));

	hipFree(d_x);
	hipFree(out_a);
	hipFree(out_b);
	return 0;
}